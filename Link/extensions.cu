#include "hip/hip_runtime.h"
/** @file
 * Contains GPU specific high-performance backend facilities,
 * which can be considered as extensions to the QuEST API.
 *
 * @author Tyson Jones
 */

#include "QuEST.h"
#include "QuEST_validation.h"

#include "errors.hpp"

#include <vector>
#include <algorithm>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>



__forceinline__ __device__ int extractBit (const int locationOfBitFromRight, const long long int theEncodedNumber) {
    return (theEncodedNumber & ( 1LL << locationOfBitFromRight )) >> locationOfBitFromRight;
}

__forceinline__ __device__ long long int flipBit(const long long int number, const int bitInd) {
    return (number ^ (1LL << bitInd));
}



__global__ void extension_addAdjointToSelfKernel(Qureg qureg) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    qreal* stateRe = qureg.deviceStateVec.real;
    qreal* stateIm = qureg.deviceStateVec.imag;
    
    // |k> = |j>|i>
    int numQubits = qureg.numQubitsRepresented;
    long long int k = thisTask;
    long long int i = k & ((1LL << numQubits)-1);
    long long int j = k >> numQubits;
    
    if (i < j) {
        // |l> = |i>|j>
        long long int l = (i << numQubits) | j;

        qreal tmp = stateRe[k] + stateRe[l];
        stateRe[k] = tmp;
        stateRe[l] = tmp;

        tmp = stateIm[k];
        stateIm[k] -= stateIm[l];
        stateIm[l] -= tmp;
    }
    else if (i == j) {
        stateRe[k] *= 2;
        stateIm[k] = 0;
    }
}

void extension_addAdjointToSelf(Qureg qureg) {
    
    validateDensityMatrQureg(qureg, "addAdjointToSelf (internal)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_applyImagFactorKernel(Qureg qureg, qreal imagFac) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    qreal* stateRe = qureg.deviceStateVec.real;
    qreal* stateIm = qureg.deviceStateVec.imag;

    // (a + b i) (fac i) = (- fac b + a fac i)
    qreal a = stateRe[thisTask];
    qreal b = stateIm[thisTask];
    stateRe[thisTask] = - imagFac * b;
    stateIm[thisTask] = + imagFac * a;
}

void extension_applyImagFactor(Qureg qureg, qreal imagFac) {
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_applyImagFactorKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, imagFac);
}



__global__ void extension_applyRealFactorKernel(Qureg qureg, qreal realFac) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;

    qureg.deviceStateVec.real[thisTask] *= realFac;
    qureg.deviceStateVec.imag[thisTask] *= realFac;
}

void extension_applyRealFactor(Qureg qureg, qreal realFac) {
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_applyRealFactorKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, realFac);
}



__global__ void extension_mixDephasingDerivKernel(Qureg qureg, int targ, qreal probDeriv) {

    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;

    long long int targShift = targ + qureg.numQubitsRepresented;
    int s = 2*(extractBit(targ, thisTask) == extractBit(targShift, thisTask)) - 1;
    qreal f = (probDeriv/2.)*(s-1);
    qureg.deviceStateVec.real[thisTask] *= f;
    qureg.deviceStateVec.imag[thisTask] *= f;
}

void extension_mixDephasingDeriv(Qureg qureg, int targ, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "Deph (derivative)");
    validateTarget(qureg, targ, "Deph (derivative)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixDephasingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targ, probDeriv);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixTwoQubitDephasingDerivKernel(Qureg qureg, int t1, int t2, qreal probDeriv) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    int t1Shift = t1 + qureg.numQubitsRepresented;
    int t2Shift = t2 + qureg.numQubitsRepresented;
    qreal c = (-2/3.) * probDeriv;
    
    long long int k = thisTask;
    int b1 = (extractBit(t1, k) == extractBit(t1Shift, k));
    int b2 = (extractBit(t2, k) == extractBit(t2Shift, k));
    qreal f =  !(b1 && b2) * c;
    qureg.deviceStateVec.real[k] *= f;
    qureg.deviceStateVec.imag[k] *= f;
}

void extension_mixTwoQubitDephasingDeriv(Qureg qureg, int t1, int t2, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "two-qubit Deph (derivative)");
    validateUniqueTargets(qureg, t1, t2, "two-qubit Deph (derivative)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixTwoQubitDephasingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, t1, t2, probDeriv);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixDepolarisingDerivKernel(Qureg qureg, int targ, qreal probDeriv) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    qreal* vecRe = qureg.deviceStateVec.real;
    qreal* vecIm = qureg.deviceStateVec.imag;
    
    long long int targShift = targ + qureg.numQubitsRepresented;
    qreal c1 = (-1/3.)*probDeriv;
    qreal c2 = 2*c1;
    
    long long int k = thisTask;
    long long int j = flipBit(flipBit(k, targ), targShift);

    if (extractBit(targ, k) == extractBit(targShift, k)) {
        if (j >= k) {
            qreal tmpRe = vecRe[k];
            qreal tmpIm = vecIm[k];
            
            vecRe[k] = c1*vecRe[k] - c1*vecRe[j];
            vecIm[k] = c1*vecIm[k] - c1*vecIm[j];

            vecRe[j] = c1*vecRe[j] - c1*tmpRe;
            vecIm[j] = c1*vecIm[j] - c1*tmpIm;
        }
    } else {
        vecRe[k] *= c2;
        vecIm[k] *= c2;
    }
}

void extension_mixDepolarisingDeriv(Qureg qureg, int targ, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "Depol (derivative)");
    validateTarget(qureg, targ, "Depol (derivative)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixDepolarisingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targ, probDeriv);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixTwoQubitDepolarisingDerivKernel(Qureg qureg, int t1, int t2, qreal c1, qreal c2) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    int numQb = qureg.numQubitsRepresented;
    int s1 = t1 + numQb;
    int s2 = t2 + numQb;
    
    qreal* vecRe = qureg.deviceStateVec.real;
    qreal* vecIm = qureg.deviceStateVec.imag;
    
    long long int k = thisTask;
    long long int j2 = flipBit(flipBit(k, t1), s1);
    long long int j3 = flipBit(flipBit(k, t2), s2);
    long long int j4 = flipBit(flipBit(j2, t2), s2);
    
    int b1 = (extractBit(t1, k) == extractBit(s1, k));
    int b2 = (extractBit(t2, k) == extractBit(s2, k));
    
    if (b2 && b1) {
        if (k < j2 && k < j3 && k < j4) {
            qreal re1 = vecRe[k];     qreal im1 = vecIm[k];
            qreal re2 = vecRe[j2];    qreal im2 = vecIm[j2];
            qreal re3 = vecRe[j3];    qreal im3 = vecIm[j3];
            qreal re4 = vecRe[j4];    qreal im4 = vecIm[j4];
            
            qreal reSum = c2 * (re1 + re2 + re3 + re4);
            qreal imSum = c2 * (im1 + im2 + im3 + im4);
            
            vecRe[k]  = c1*vecRe[k]  + reSum;
            vecRe[j2] = c1*vecRe[j2] + reSum;
            vecRe[j3] = c1*vecRe[j3] + reSum;
            vecRe[j4] = c1*vecRe[j4] + reSum;

            vecIm[k]  = c1*vecIm[k]  + imSum;
            vecIm[j2] = c1*vecIm[j2] + imSum;
            vecIm[j3] = c1*vecIm[j3] + imSum;
            vecIm[j4] = c1*vecIm[j4] + imSum;
        }
    } else {
        vecRe[k] *= c1;
        vecIm[k] *= c1;
    }
}

void extension_mixTwoQubitDepolarisingDeriv(Qureg qureg, int t1, int t2, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "two-qubit Depol (derivative)");
    validateUniqueTargets(qureg, t1, t2, "two-qubit Depol (derivative)");
    
    qreal c1 = (-8/15.)*probDeriv;
    qreal c2 = ( 2/15.)*probDeriv;
    
    // 12 of every 16 amplitudes merely scale,
    // the remaining 4 are mixed
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixTwoQubitDepolarisingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, t1, t2, c1, c2);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixDampingDerivKernel(Qureg qureg, int targ, qreal c1, qreal c2) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    int conjTarg = targ + qureg.numQubitsRepresented;
    qreal* vecRe = qureg.deviceStateVec.real;
    qreal* vecIm = qureg.deviceStateVec.imag;
    
    long long int i = thisTask;
    int b1 = extractBit(targ, i);
    int b2 = extractBit(conjTarg, i);

    /* easy to refactor without warp divergence, albeit with the same 
     * (possibly even worse) memory bottleneck due to non-local memory 
     * modification in the last condition 
     */
    if (b2 == 0 && b1 == 1) {
        vecRe[i] *= c2;
        vecIm[i] *= c2;
    } else if (b2 == 1 && b1 == 0) {
        vecRe[i] = 0;
        vecIm[i] = 0;
    } else if (b1 == 1 && b2 == 1){
        vecRe[i] *= - c1;
        vecIm[i] *= - c1;
        long long int j = flipBit(flipBit(i, targ), conjTarg);
        vecRe[j] = - vecRe[i];
        vecIm[j] = - vecIm[i];
    }
}

void extension_mixDampingDeriv(Qureg qureg, int targ, qreal prob, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "Damp (derivative)");
    validateTarget(qureg, targ, "Damp (derivative)");
    
    qreal c1 = probDeriv/2.;
    qreal c2 = - c1 / sqrt(1. - prob);
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixDampingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targ, c1, c2);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}

__global__ void local_validateShadowPaulisKernel(int* invalid, int numQb, unsigned long long  numTotalPaulis, int* pauliCodes, int* pauliTargs) {
    long long int k = blockIdx.x*blockDim.x + threadIdx.x;
    if (k >= numTotalPaulis) return;
    
    if (pauliCodes[k] < 0 || pauliCodes[k] > 3 || pauliTargs[k] < 0 || pauliTargs[k] >= numQb)
        *invalid = 1;
}

__global__ void local_validateShadowSampsKernel(int* invalid, unsigned long long numTotalSampVals, int* sampleBases, int* sampleOutcomes) {
    long long int k = blockIdx.x*blockDim.x + threadIdx.x;
    if (k >= numTotalSampVals) return;
    
    if (sampleBases[k] < 1 || sampleBases[k] > 3 || sampleOutcomes[k] < 0 || sampleOutcomes[k] > 1)
        *invalid = 1;
}

__global__ void local_prepareShadowPauliBitseqsKernel(
    unsigned long long* pauliBitseqs, unsigned long long* pauliTargBitseqs, unsigned long long* outcomeTargBitseqs,
    long numProds, int* numPaulisPerProd, long* pauliIndOffset, int* pauliCodes, int* pauliTargs
) {
    long long int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= numProds) return;
    
    int offset = pauliIndOffset[i];
    int numPaulisInProd = numPaulisPerProd[i];
    
    unsigned long long seqPaulis = 00ULL;      // length 2*numQb
    unsigned long long seqPauliTargs = 00ULL;  // length 2*numQb
    unsigned long long seqOutcomeTargs = 0ULL; // length numQb
    
    for (int p=0; p<numPaulisInProd; p++) {
        int j = offset + p;
        seqPaulis |= (pauliCodes[j] << 2*pauliTargs[j]);
        seqPauliTargs |= (3ULL << 2*pauliTargs[j]);
        seqOutcomeTargs |= (1ULL << pauliTargs[j]);
    }
    
    pauliBitseqs[i] = seqPaulis;
    pauliTargBitseqs[i] = seqPauliTargs;
    outcomeTargBitseqs[i] = seqOutcomeTargs;
}

__global__ void local_prepareShadowSampleBitseqsKernel(
    unsigned long long* baseBitseqs, unsigned long long* outcomeBitseqs,
    int numQb, long numSamples, int* sampleBases, int* sampleOutcomes
) {
    long long int s = blockIdx.x*blockDim.x + threadIdx.x;
    if (s >= numSamples) return;
    
    long offset = numQb*s;
    unsigned long long seqBases = 00ULL;   // length 2*numQb
    unsigned long long seqOuts = 0ULL;     // length numQb

    for (int q=numQb-1; q>=0; q--) {
        int ind = offset + q;
        seqBases = (seqBases << 2) | sampleBases[ind];
        seqOuts = (seqOuts << 1) | sampleOutcomes[ind];
    }
        
    baseBitseqs[s] = seqBases;
    outcomeBitseqs[s] = seqOuts;
}

#define MAX_NUM_SHADOW_BATCHES 200
    
__global__ void extension_calcExpecPauliProdsFromClassicalShadowKernel(
    qreal* prodExpecVals,
    int numProds, int numSamples, int numBatches,
    int* numPaulisPerProd,
    unsigned long long *baseBitseqs, unsigned long long *pauliTargBitseqs, 
    unsigned long long *pauliBitseqs, unsigned long long *outcomeBitseqs, 
    unsigned long long *outcomeTargBitseqs
) {
    // parallel evauate the expected values of each pauli product, modifying output array prodExpecVals.
    long long int p = blockIdx.x*blockDim.x + threadIdx.x;
    if (p >= numProds) return;
    
    // divide this thread's job into batches 
    qreal batchVals[MAX_NUM_SHADOW_BATCHES];
    long numSampsPerBatch = (long) ceil(numSamples/(qreal) numBatches);
    int numProdPaulis = numPaulisPerProd[p];
    
    // populate batch values
    for (int b=0; b<numBatches; b++) {
        
        // batch indivisibility may mean final batch has fewer samples than others
        bool isFinalBatch = (b == (numBatches-1));
        long batchSize = isFinalBatch * (numSamples-(numBatches-1)*numSampsPerBatch) + (1-isFinalBatch) * numSampsPerBatch;
        
        long val = 0;
        for (long i=0; i<batchSize; i++) {
            long s = b*batchSize + i;
            
            // determine whether this sample matches the pauli product
            int match = (baseBitseqs[s] & pauliTargBitseqs[p]) == pauliBitseqs[p];
            
            // obtain the outcomes of only the targeted qubits (rest forced to 0)
            unsigned long long targOuts = (outcomeBitseqs[s] & outcomeTargBitseqs[p]);

            // bit-twiddling hack to determine parity of number of 1s in targOuts
            // https://graphics.stanford.edu/~seander/bithacks.html#ParityMultiply
            targOuts ^= targOuts >> 1;
            targOuts ^= targOuts >> 2;
            targOuts = (targOuts & 0x1111111111111111UL) * 0x1111111111111111UL;
            int par = (targOuts >> 60) & 1;
            
            // contribute sample if match (without branching)
            val += match * (1-2*par);
        }
        
        batchVals[b] = val / (qreal) batchSize;
    }
    
    qreal fac = pow((qreal) 3., (qreal) numProdPaulis); 
    
    // choose the median of the batch values
    thrust::sort(thrust::seq, batchVals, &(batchVals[numBatches]));
    if (numBatches % 2)
        prodExpecVals[p] = fac * batchVals[numBatches/2];
    else 
        prodExpecVals[p] = fac * .5 * (batchVals[numBatches/2] + batchVals[numBatches/2 + 1]);
}

void extension_calcExpecPauliProdsFromClassicalShadow(
    std::vector<qreal> &prodExpecVals, long numProds,
    int* sampleBases, int* sampleOutcomes, int numQb, long numSamples,
    int* pauliCodes, int* pauliTargs, int* numPaulisPerProd,
    int numBatches
) {
    // early numBatches validation, since we must be more strict than the CPU variant
    if (numBatches > MAX_NUM_SHADOW_BATCHES)
        throw QuESTException("", "The maximum number of batches permitted in GPU mode is " + std::to_string(MAX_NUM_SHADOW_BATCHES));
    
    // serial array encode: O(numProds)
    std::vector<long> pauliIndOffset(numProds);
    pauliIndOffset[0] = 0;
    for (int i=1; i<numProds; i++)
        pauliIndOffset[i] = pauliIndOffset[i-1] + numPaulisPerProd[i-1];
        
    // prepare device copy of output structure
    qreal* d_prodExpecVals;
    size_t memExpecVals = numProds * sizeof *d_prodExpecVals;
    hipMalloc(&d_prodExpecVals, memExpecVals);
        
    // prepare device copies of input structures
    unsigned long long numTotalPaulis = pauliIndOffset[numProds-1] + numPaulisPerProd[numProds-1];
    size_t memSamps = numSamples*numQb * sizeof(int);
    size_t memPaulis = numTotalPaulis * sizeof (int);
    size_t memProds = numProds * sizeof (int);
    int* d_sampleBases;         hipMalloc(&d_sampleBases, memSamps);       hipMemcpy(d_sampleBases, sampleBases, memSamps, hipMemcpyHostToDevice);
    int* d_sampleOutcomes;      hipMalloc(&d_sampleOutcomes, memSamps);    hipMemcpy(d_sampleOutcomes, sampleOutcomes, memSamps, hipMemcpyHostToDevice);
    int* d_pauliCodes;          hipMalloc(&d_pauliCodes, memPaulis);       hipMemcpy(d_pauliCodes, pauliCodes, memPaulis, hipMemcpyHostToDevice);
    int* d_pauliTargs;          hipMalloc(&d_pauliTargs, memPaulis);       hipMemcpy(d_pauliTargs, pauliTargs, memPaulis, hipMemcpyHostToDevice);
    int* d_numPaulisPerProd;    hipMalloc(&d_numPaulisPerProd, memProds);  hipMemcpy(d_numPaulisPerProd, numPaulisPerProd, memProds, hipMemcpyHostToDevice);
    
    // prepare device working structures
    size_t memOffset = numProds * sizeof(long);
    long *d_pauliIndOffset; 
    hipMalloc(&d_pauliIndOffset, memOffset);
    hipMemcpy(d_pauliIndOffset, pauliIndOffset.data(), memOffset, hipMemcpyHostToDevice);
    
    memProds = numProds * sizeof(unsigned long long);
    memSamps = numSamples * sizeof(unsigned long long);
    unsigned long long *d_pauliBitseqs;         hipMalloc(&d_pauliBitseqs, memProds);
    unsigned long long *d_pauliTargBitseqs;     hipMalloc(&d_pauliTargBitseqs, memProds);
    unsigned long long *d_outcomeTargBitseqs;   hipMalloc(&d_outcomeTargBitseqs, memProds);
    unsigned long long *d_baseBitseqs;          hipMalloc(&d_baseBitseqs, memSamps);
    unsigned long long *d_outcomeBitseqs;       hipMalloc(&d_outcomeBitseqs, memSamps);
    
    int bs = 128; // blocksize for GPU parallelisation
    
    // perform validation 
    int invalid = 0;
    int *d_invalid;
    hipMalloc(&d_invalid, sizeof(int));
    hipMemcpy(d_invalid, &invalid, sizeof(int), hipMemcpyHostToDevice); 
    local_validateShadowPaulisKernel<<<ceil(numTotalPaulis/(qreal)bs), bs>>>(
        d_invalid, numQb, numTotalPaulis, d_pauliCodes, d_pauliTargs);
    local_validateShadowSampsKernel<<<ceil(numSamples*numQb/(qreal)bs), bs>>>(
        d_invalid, numSamples*numQb, d_sampleBases, d_sampleOutcomes);
    hipMemcpy(&invalid, d_invalid, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_invalid);
    
    if (!invalid) {
        
        // prepare bit sequences
        local_prepareShadowPauliBitseqsKernel<<<ceil(numProds/(qreal)bs), bs>>>(
            d_pauliBitseqs, d_pauliTargBitseqs, d_outcomeTargBitseqs,
            numProds, d_numPaulisPerProd, d_pauliIndOffset, d_pauliCodes, d_pauliTargs);
        local_prepareShadowSampleBitseqsKernel<<<ceil(numSamples/(qreal)bs), bs>>>(
            d_baseBitseqs, d_outcomeBitseqs,
            numQb, numSamples, d_sampleBases, d_sampleOutcomes);
            
        // evaluate shadow expec values
        extension_calcExpecPauliProdsFromClassicalShadowKernel<<<ceil(numSamples/(qreal)bs), bs>>>(
            d_prodExpecVals,
            numProds, numSamples, numBatches, 
            d_numPaulisPerProd, d_baseBitseqs, d_pauliTargBitseqs, 
            d_pauliBitseqs, d_outcomeBitseqs, d_outcomeTargBitseqs);
            
        // copy expec vals back to RAM 
        hipMemcpy(prodExpecVals.data(), d_prodExpecVals, memExpecVals, hipMemcpyDeviceToHost);
    }

    // clean-up
    hipFree(d_prodExpecVals);
    hipFree(d_sampleOutcomes);
    hipFree(d_pauliCodes);
    hipFree(d_pauliTargs);
    hipFree(d_numPaulisPerProd);
    hipFree(d_pauliIndOffset);
    hipFree(d_pauliBitseqs);
    hipFree(d_pauliTargBitseqs);
    hipFree(d_outcomeTargBitseqs);
    hipFree(d_baseBitseqs);
    hipFree(d_outcomeBitseqs);
    
    if (invalid)
        throw QuESTException("", "The input classical shadow, or the Pauli products, were invalid.");
}

