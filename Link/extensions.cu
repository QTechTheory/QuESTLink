#include "hip/hip_runtime.h"
/** @file
 * Contains GPU specific high-performance backend facilities,
 * which can be considered as extensions to the QuEST API.
 *
 * @author Tyson Jones
 */

#include "QuEST.h"
#include "QuEST_validation.h"



__forceinline__ __device__ int extractBit (const int locationOfBitFromRight, const long long int theEncodedNumber) {
    return (theEncodedNumber & ( 1LL << locationOfBitFromRight )) >> locationOfBitFromRight;
}

__forceinline__ __device__ long long int flipBit(const long long int number, const int bitInd) {
    return (number ^ (1LL << bitInd));
}



__global__ void extension_addAdjointToSelfKernel(Qureg qureg) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    qreal* stateRe = qureg.deviceStateVec.real;
    qreal* stateIm = qureg.deviceStateVec.imag;
    
    // |k> = |j>|i>
    int numQubits = qureg.numQubitsRepresented;
    long long int k = thisTask;
    long long int i = k & ((1LL << numQubits)-1);
    long long int j = k >> numQubits;
    
    if (i < j) {
        // |l> = |i>|j>
        long long int l = (i << numQubits) | j;

        qreal tmp = stateRe[k] + stateRe[l];
        stateRe[k] = tmp;
        stateRe[l] = tmp;

        tmp = stateIm[k];
        stateIm[k] -= stateIm[l];
        stateIm[l] -= tmp;
    }
    else if (i == j) {
        stateRe[k] *= 2;
        stateIm[k] = 0;
    }
}

void extension_addAdjointToSelf(Qureg qureg) {
    
    validateDensityMatrQureg(qureg, "addAdjointToSelf (internal)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_applyImagFactorKernel(Qureg qureg, qreal imagFac) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    qreal* stateRe = qureg.deviceStateVec.real;
    qreal* stateIm = qureg.deviceStateVec.imag;

    // (a + b i) (fac i) = (- fac b + a fac i)
    qreal a = stateRe[thisTask];
    qreal b = stateIm[thisTask];
    stateRe[thisTask] = - imagFac * b;
    stateIm[thisTask] = + imagFac * a;
}

void extension_applyImagFactor(Qureg qureg, qreal imagFac) {
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_applyImagFactorKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, imagFac);
}



__global__ void extension_mixDephasingDerivKernel(Qureg qureg, int targ, qreal probDeriv) {

    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;

    long long int targShift = targ + qureg.numQubitsRepresented;
    int s = 2*(extractBit(targ, thisTask) == extractBit(targShift, thisTask)) - 1;
    qreal f = (probDeriv/2.)*(s-1);
    qureg.deviceStateVec.real[thisTask] *= f;
    qureg.deviceStateVec.imag[thisTask] *= f;
}

void extension_mixDephasingDeriv(Qureg qureg, int targ, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "Deph (derivative)");
    validateTarget(qureg, targ, "Deph (derivative)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixDephasingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targ, probDeriv);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixTwoQubitDephasingDerivKernel(Qureg qureg, int t1, int t2, qreal probDeriv) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    int t1Shift = t1 + qureg.numQubitsRepresented;
    int t2Shift = t2 + qureg.numQubitsRepresented;
    qreal c = (-2/3.) * probDeriv;
    
    long long int k = thisTask;
    int b1 = (extractBit(t1, k) == extractBit(t1Shift, k));
    int b2 = (extractBit(t2, k) == extractBit(t2Shift, k));
    qreal f =  !(b1 && b2) * c;
    qureg.deviceStateVec.real[k] *= f;
    qureg.deviceStateVec.imag[k] *= f;
}

void extension_mixTwoQubitDephasingDeriv(Qureg qureg, int t1, int t2, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "two-qubit Deph (derivative)");
    validateUniqueTargets(qureg, t1, t2, "two-qubit Deph (derivative)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixTwoQubitDephasingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, t1, t2, probDeriv);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixDepolarisingDerivKernel(Qureg qureg, int targ, qreal probDeriv) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    qreal* vecRe = qureg.deviceStateVec.real;
    qreal* vecIm = qureg.deviceStateVec.imag;
    
    long long int targShift = targ + qureg.numQubitsRepresented;
    qreal c1 = (-1/3.)*probDeriv;
    qreal c2 = 2*c1;
    
    long long int k = thisTask;
    long long int j = flipBit(flipBit(k, targ), targShift);

    if (extractBit(targ, k) == extractBit(targShift, k)) {
        if (j >= k) {
            qreal tmpRe = vecRe[k];
            qreal tmpIm = vecIm[k];
            
            vecRe[k] = c1*vecRe[k] - c1*vecRe[j];
            vecIm[k] = c1*vecIm[k] - c1*vecIm[j];

            vecRe[j] = c1*vecRe[j] - c1*tmpRe;
            vecIm[j] = c1*vecIm[j] - c1*tmpIm;
        }
    } else {
        vecRe[k] *= c2;
        vecIm[k] *= c2;
    }
}

void extension_mixDepolarisingDeriv(Qureg qureg, int targ, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "Depol (derivative)");
    validateTarget(qureg, targ, "Depol (derivative)");
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixDepolarisingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targ, probDeriv);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixTwoQubitDepolarisingDerivKernel(Qureg qureg, int t1, int t2, qreal c1, qreal c2) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    int numQb = qureg.numQubitsRepresented;
    int s1 = t1 + numQb;
    int s2 = t2 + numQb;
    
    qreal* vecRe = qureg.deviceStateVec.real;
    qreal* vecIm = qureg.deviceStateVec.imag;
    
    long long int k = thisTask;
    long long int j2 = flipBit(flipBit(k, t1), s1);
    long long int j3 = flipBit(flipBit(k, t2), s2);
    long long int j4 = flipBit(flipBit(j2, t2), s2);
    
    int b1 = (extractBit(t1, k) == extractBit(s1, k));
    int b2 = (extractBit(t2, k) == extractBit(s2, k));
    
    if (b2 && b1) {
        if (k < j2 && k < j3 && k < j4) {
            qreal re1 = vecRe[k];     qreal im1 = vecIm[k];
            qreal re2 = vecRe[j2];    qreal im2 = vecIm[j2];
            qreal re3 = vecRe[j3];    qreal im3 = vecIm[j3];
            qreal re4 = vecRe[j4];    qreal im4 = vecIm[j4];
            
            qreal reSum = c2 * (re1 + re2 + re3 + re4);
            qreal imSum = c2 * (im1 + im2 + im3 + im4);
            
            vecRe[k]  = c1*vecRe[k]  + reSum;
            vecRe[j2] = c1*vecRe[j2] + reSum;
            vecRe[j3] = c1*vecRe[j3] + reSum;
            vecRe[j4] = c1*vecRe[j4] + reSum;

            vecIm[k]  = c1*vecIm[k]  + imSum;
            vecIm[j2] = c1*vecIm[j2] + imSum;
            vecIm[j3] = c1*vecIm[j3] + imSum;
            vecIm[j4] = c1*vecIm[j4] + imSum;
        }
    } else {
        vecRe[k] *= c1;
        vecIm[k] *= c1;
    }
}

void extension_mixTwoQubitDepolarisingDeriv(Qureg qureg, int t1, int t2, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "two-qubit Depol (derivative)");
    validateUniqueTargets(qureg, t1, t2, "two-qubit Depol (derivative)");
    
    qreal c1 = (-8/15.)*probDeriv;
    qreal c2 = ( 2/15.)*probDeriv;
    
    // 12 of every 16 amplitudes merely scale,
    // the remaining 4 are mixed
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixTwoQubitDepolarisingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, t1, t2, c1, c2);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}



__global__ void extension_mixDampingDerivKernel(Qureg qureg, int targ, qreal c1, qreal c2) {
    
    // each thread modifies one value (blegh)
    long long int numTasks = qureg.numAmpsPerChunk;
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask >= numTasks) return;
    
    int conjTarg = targ + qureg.numQubitsRepresented;
    qreal* vecRe = qureg.deviceStateVec.real;
    qreal* vecIm = qureg.deviceStateVec.imag;
    
    long long int i = thisTask;
    int b1 = extractBit(targ, i);
    int b2 = extractBit(conjTarg, i);

    /* easy to refactor without warp divergence, albeit with the same 
     * (possibly even worse) memory bottleneck due to non-local memory 
     * modification in the last condition 
     */
    if (b2 == 0 && b1 == 1) {
        vecRe[i] *= c2;
        vecIm[i] *= c2;
    } else if (b2 == 1 && b1 == 0) {
        vecRe[i] = 0;
        vecIm[i] = 0;
    } else if (b1 == 1 && b2 == 1){
        vecRe[i] *= - c1;
        vecIm[i] *= - c1;
        long long int j = flipBit(flipBit(i, targ), conjTarg);
        vecRe[j] = - vecRe[i];
        vecIm[j] = - vecIm[i];
    }
}

void extension_mixDampingDeriv(Qureg qureg, int targ, qreal prob, qreal probDeriv) {
    
    validateDensityMatrQureg(qureg, "Damp (derivative)");
    validateTarget(qureg, targ, "Damp (derivative)");
    
    qreal c1 = probDeriv/2.;
    qreal c2 = - c1 / sqrt(1. - prob);
    
    int threadsPerCUDABlock = 128;
    int CUDABlocks = ceil(qureg.numAmpsPerChunk/ (qreal) threadsPerCUDABlock);
    extension_mixDampingDerivKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targ, c1, c2);
    extension_addAdjointToSelfKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg);
}
